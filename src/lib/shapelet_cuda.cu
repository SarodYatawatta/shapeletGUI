#include "hip/hip_runtime.h"
/* FST - a Fast Shapelet Transformer
 *
   Copyright (C) 2006-2024 Sarod Yatawatta <sarod@users.sf.net>  
 This program is free software; you can redistribute it and/or modify
 it under the terms of the GNU General Public License as published by
 the Free Software Foundation; either version 2 of the License, or
 (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software
 Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
 $Id$
*/

#include <hip/hip_runtime.h>
#include "shapelet.h"

#define DEFAULT_TH_PER_BK 64

/* Hermite polynomial, non recursive version */
__device__ float
H_e(float x, int n) {
  if(n==0) return 1.0f;
  if(n==1) return 2.0f*x;
  /* else iterate */
  float Hn_1,Hn,Hnp1;
  Hn_1=1.0f;
  Hn=2.0f*x;
  int ci;
  for (ci=1; ci<n; ci++) {
    Hnp1=2.0f*x*Hn-2.0f*((float)ci)*Hn_1;
    Hn_1=Hn;
    Hn=Hnp1;
  }

  return Hn;
}

__global__ void 
kernel_calculate_shapelet_lm(float *Ad,float *xd,float *yd,float *fact,float beta,int N,int n0, int startpix, int endpix) {
  /* pixel 0..N, n+startpix */
  unsigned int n = threadIdx.x + blockDim.x*blockIdx.x + startpix;
  /* mode 0...n0^2 */
  unsigned int mode = threadIdx.y + blockDim.y*blockIdx.y;
  /* separate mode to n1,n2 */
  unsigned int n1=mode%n0;
  unsigned int n2=mode/n0;

  if (n<=endpix && n1<n0 && n2<n0) {
   float xx=xd[n]/beta;
   float yy=yd[n]/beta;

   Ad[n+mode*N]=H_e(xx,n1)/sqrtf(powf(2.0f,(float)n1+1)*fact[n1])*expf(-0.5f*xx*xx)
    *H_e(yy,n2)/sqrtf(powf(2.0f,(float)n2+1)*fact[n2])*expf(-0.5f*yy*yy);

  }
}

extern "C" {

static void
checkCudaError(hipError_t err, const char *file, int line)
{
    if(!err)
        return;
    fprintf(stderr,"GPU (CUDA): %s %s %d\n", hipGetErrorString(err),file,line);
    exit(EXIT_FAILURE);
}

int 
calculate_mode_vectors_cuda(double *x, double *y, int N,  double beta, int n0, double **Av) {

  hipError_t error;
  error = hipGetLastError();

  /* set up factorial array */
  float *fact;
  if ((fact=(float*)calloc((size_t)(n0),sizeof(float)))==0) {
    fprintf(stderr,"%s: %d: no free memory\n",__FILE__,__LINE__);
    exit(1);
  }
  fact[0]=1.0f;
  for (int ci=1; ci<(n0); ci++) {
    fact[ci]=float(ci)*fact[ci-1];
  }

  float *Ad=0;
  float *factd=0;
  float *xd=0;
  float *yd=0;
  /* setup device memory */
  error=hipMalloc((void**)&Ad, (size_t)(N*(n0)*(n0))*sizeof(float));
  checkCudaError(error,__FILE__,__LINE__);
  error=hipMalloc((void**)&factd, (size_t)(n0)*sizeof(float));
  checkCudaError(error,__FILE__,__LINE__);
  error=hipMemcpy(factd,fact,(size_t)(n0)*sizeof(float),hipMemcpyHostToDevice);
  checkCudaError(error,__FILE__,__LINE__);
  error=hipMalloc((void**)&xd, (size_t)(N)*sizeof(float));
  checkCudaError(error,__FILE__,__LINE__);
  error=hipMalloc((void**)&yd, (size_t)(N)*sizeof(float));
  checkCudaError(error,__FILE__,__LINE__);
  float *xf,*yf;
  if ((xf=(float*)calloc((size_t)(N),sizeof(float)))==0) {
    fprintf(stderr,"%s: %d: no free memory\n",__FILE__,__LINE__);
    exit(1);
  }
  if ((yf=(float*)calloc((size_t)(N),sizeof(float)))==0) {
    fprintf(stderr,"%s: %d: no free memory\n",__FILE__,__LINE__);
    exit(1);
  }
  for (int ci=0; ci<N; ci++) {
    xf[ci]=(float)x[ci];
    yf[ci]=(float)y[ci];
  }
  error=hipMemcpy(xd,xf,(size_t)(N)*sizeof(float),hipMemcpyHostToDevice);
  checkCudaError(error,__FILE__,__LINE__);
  error=hipMemcpy(yd,yf,(size_t)(N)*sizeof(float),hipMemcpyHostToDevice);
  checkCudaError(error,__FILE__,__LINE__);

  free(xf);
  free(yf);

  int ThreadsPerBlock=DEFAULT_TH_PER_BK;
  /* if pixels x modes / threads > 16384, split the calculations into smaller chunks */
  int MAX_BLOCKS=16384;
  if (n0*n0*N/ThreadsPerBlock < MAX_BLOCKS) {
    /* 2D grid of threads: x dim-> pixel (N), y dim-> shapelet mode (n0*n0) */
    dim3 grid(1, 1, 1);
    grid.x = (int)ceilf(N / (float)ThreadsPerBlock);
    grid.y = n0*n0;
    int startpix=0;
    int endpix=N-1;

    kernel_calculate_shapelet_lm<<<grid,ThreadsPerBlock>>>(Ad,xd,yd,factd,(float)beta,N,n0,startpix,endpix);
  } else {
    /* divide N pixels into blocks so that
       pixels * modes / threads < 16384 */
    int n_runs=(int)ceilf(ceilf((N*n0*n0)/(float)ThreadsPerBlock)/(float)MAX_BLOCKS);
    int pix_per_run=N/n_runs;
    for (int run=0; run<n_runs; run++) {
      int startpix=run*pix_per_run;
      int endpix=(run+1)*pix_per_run-1;
      if (endpix>N-1) { endpix=N-1; }
      dim3 grid(1, 1, 1);
      grid.x = (int)ceilf((endpix-startpix+1) / (float)ThreadsPerBlock);
      grid.y = n0*n0;

      kernel_calculate_shapelet_lm<<<grid,ThreadsPerBlock>>>(Ad,xd,yd,factd,(float)beta,N,n0,startpix,endpix);
    }
  }
  hipDeviceSynchronize();
  error = hipGetLastError();
  checkCudaError(error,__FILE__,__LINE__);

  free(fact);
  error=hipFree(factd);
  checkCudaError(error,__FILE__,__LINE__);
  error=hipFree(xd);
  checkCudaError(error,__FILE__,__LINE__);
  error=hipFree(yd);
  checkCudaError(error,__FILE__,__LINE__);

  float *Af=0;
  if ((Af=(float*)calloc((size_t)(N*(n0)*(n0)),sizeof(float)))==0) {
    fprintf(stderr,"%s: %d: no free memory\n",__FILE__,__LINE__);
    exit(1);
  }
  error=hipMemcpy(Af,Ad,(size_t)(N*(n0)*(n0))*sizeof(float),hipMemcpyDeviceToHost);
  checkCudaError(error,__FILE__,__LINE__);
  error=hipFree(Ad);
  checkCudaError(error,__FILE__,__LINE__);

  if ((*Av=(double*)calloc((size_t)(N*(n0)*(n0)),sizeof(double)))==0) {
    fprintf(stderr,"%s: %d: no free memory\n",__FILE__,__LINE__);
    exit(1);
  }
  for (int ci=0; ci<N*n0*n0; ci++) {
    (*Av)[ci]=(double)Af[ci];
  }

  free(Af);
  return 0;

}

}
